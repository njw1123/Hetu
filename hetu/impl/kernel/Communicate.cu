#include "hetu/core/ndarray.h"
#include "hetu/core/stream.h"
#include "hetu/impl/communication/nccl_comm_group.h"
#include "hetu/impl/communication/torch_nccl_comm_group.h"
#include "hetu/impl/utils/common_utils.h"
#include "hetu/impl/stream/CUDAStream.h"

#include <thread>

namespace hetu {
namespace impl {

using namespace hetu::impl::comm;

void AllReduceCuda(const NDArray& input, NDArray& output, ReductionType red_type,
                   const DeviceGroup& device_group, const Stream& stream) {
  auto ranks = DeviceGroupToWorldRanks(device_group);
  auto& comm_group = NCCLCommunicationGroup::GetOrCreate(ranks, stream);
  comm_group->AllReduce(input, output, red_type);
  NDArray::MarkUsedBy({input, output}, stream);  
}

void AllGatherCuda(const NDArray& input, NDArray& output,
                   const DeviceGroup& device_group, int32_t gather_dim, const Stream& stream) {
  auto ranks = DeviceGroupToWorldRanks(device_group);
  auto& comm_group = NCCLCommunicationGroup::GetOrCreate(ranks, stream);
  comm_group->AllGather(input, output, gather_dim); 
  NDArray::MarkUsedBy({input, output}, stream);                   
}

void ReduceScatterCuda(const NDArray& input, NDArray& output, ReductionType red_type,
                   const DeviceGroup& device_group, int32_t scatter_dim, const Stream& stream) {
  auto ranks = DeviceGroupToWorldRanks(device_group);
  auto& comm_group = NCCLCommunicationGroup::GetOrCreate(ranks, stream);
  comm_group->ReduceScatter(input, output, scatter_dim, red_type);
  NDArray::MarkUsedBy({input, output}, stream);  
}

void P2PSendCuda(const NDArray& data, const Device& dst, const std::vector<int>& comm_group_ranks, const Stream& stream) {
  auto dst_rank = DeviceToWorldRank(dst);
  auto& comm_group = NCCLCommunicationGroup::GetOrCreate(comm_group_ranks, stream);
  comm_group->Send(data, dst_rank);
  NDArray::MarkUsedBy({data}, stream);
}

void P2PRecvCuda(NDArray& data, const Device& src, const std::vector<int>& comm_group_ranks, const Stream& stream) {
  auto src_rank = DeviceToWorldRank(src);
  auto& comm_group = NCCLCommunicationGroup::GetOrCreate(comm_group_ranks, stream);
  comm_group->Recv(data, src_rank);
  NDArray::MarkUsedBy({data}, stream);
}

void BatchedISendIRecvCuda(const NDArrayList& send_datas, 
  const std::vector<Device>& dsts, NDArrayList& recv_datas, 
  const std::vector<Device>& srcs, const std::vector<Device>& comm_deivces, 
  const Stream& stream) {
  std::vector<int> ranks(comm_deivces.size());
  std::transform(comm_deivces.begin(), comm_deivces.end(), ranks.begin(), [&](const Device& device) { return DeviceToWorldRank(device); });
  std::sort(ranks.begin(), ranks.end());
  auto& comm_group = NCCLCommunicationGroup::GetOrCreate(ranks, stream);
  std::vector<CommTask> tasks;
  tasks.reserve(send_datas.size() + recv_datas.size());
  for (int i = 0; i < send_datas.size(); i++) {
    tasks.push_back(comm_group->ISend(send_datas[i], DeviceToWorldRank(dsts[i])));
  }
  for (int i = 0; i < recv_datas.size(); i++) {
    tasks.push_back(comm_group->IRecv(recv_datas[i], DeviceToWorldRank(srcs[i])));
  }
  comm_group->BatchedISendIRecv(tasks);
  NDArray::MarkUsedBy(send_datas, stream);
  NDArray::MarkUsedBy(recv_datas, stream);
}

void BroadcastCommCuda(NDArray& data, int broadcaster,
                       const DeviceGroup& device_group, const Stream& stream) {
  auto ranks = DeviceGroupToWorldRanks(device_group);
  auto& comm_group = NCCLCommunicationGroup::GetOrCreate(ranks, stream);
  comm_group->Broadcast(data, broadcaster);
  NDArray::MarkUsedBy({data}, stream);
}

void ReduceCommCuda(const NDArray& input, NDArray& output, int reducer,
                 const DeviceGroup& device_group, const Stream& stream) {
  auto ranks = DeviceGroupToWorldRanks(device_group);
  auto& comm_group = NCCLCommunicationGroup::GetOrCreate(ranks, stream);
  comm_group->Reduce(input, output, reducer);
  NDArray::MarkUsedBy({input, output}, stream);  
}

void GatherCuda(const NDArray& input, NDArray& output, int gatherer,
                   const DeviceGroup& device_group, const Stream& stream) {
  auto ranks = DeviceGroupToWorldRanks(device_group);
  auto& comm_group = NCCLCommunicationGroup::GetOrCreate(ranks, stream);
  comm_group->Gather(input, output, gatherer);
  NDArray::MarkUsedBy({input, output}, stream);  
}

void ScatterCuda(const NDArray& input, NDArray& output, int scatterer,
                   const DeviceGroup& device_group, const Stream& stream) {
  auto ranks = DeviceGroupToWorldRanks(device_group);
  auto& comm_group = NCCLCommunicationGroup::GetOrCreate(ranks, stream);
  comm_group->Scatter(input, output, scatterer);
  NDArray::MarkUsedBy({input, output}, stream);  
}

} // namespace impl
} // namespace hetu
